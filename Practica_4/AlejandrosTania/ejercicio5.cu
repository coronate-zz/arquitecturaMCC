#include "hip/hip_runtime.h"
/*
 * Plantilla para la multiplicación de matrices
 * con memoria compartida
 * Jose Incera. Adaptado del código
 * de Robert Hochberg
 * Abril 2016
 *
 * Based nearly entirely on the code from the CUDA C Programming Guide
 */


#include <stdio.h>
#include <sys/time.h>
#include <sys/resource.h>

// Estructura Matriz.
typedef struct{
    int nRen;
    int nCol;
    int *elementos;
    int salto; // stride para recorrer columnas
} Matriz;

// dimensión de un bloque
// El tamaño es TAM_BLOQUE * TAM_BLOQUE
#define TAM_BLOQUE 16

// Prototipo de función
__global__ void MatMultKernel(const Matriz, const Matriz, Matriz);

// Por facilidad, las dimensiones de la matriz son múltiplos de TAM_BLOQUE
void MatMult(const Matriz A, const Matriz B, Matriz C) {
    
    // Carga A y B en memoria GPU
    Matriz d_A;
    d_A.nRen = d_A.salto = A.nRen;
    d_A.nCol = A.nCol;
    size_t tam= A.nRen * A.nCol * sizeof(int);
    
    hipError_t err = hipMalloc((void **)&(d_A.elementos),tam);  //  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    hipMemcpy(d_A.elementos,A.elementos,tam,hipMemcpyHostToDevice); //  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    
    Matriz d_B;
    d_B.nRen = d_B.salto = B.nRen;
    d_B.nCol = B.nCol;
    tam= B.nRen * B.nCol * sizeof(int);
    
    hipMalloc((void **)&(d_B.elementos),tam);  //  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    hipMemcpy(d_B.elementos,B.elementos,tam,hipMemcpyHostToDevice); //  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    
    // Asigna espacio para C en GPU
    Matriz d_C;
    d_C.nRen = d_C.salto = C.nRen;
    d_C.nCol = C.nCol;
    tam = C.nRen * C.nCol * sizeof(int);
    hipMalloc((void **)&(d_C.elementos),tam);  //  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    
    // Llama al kernel
    dim3 dimBlock(TAM_BLOQUE, TAM_BLOQUE);
    dim3 dimGrid(B.nRen / dimBlock.x, A.nCol / dimBlock.y);
    
    //  Descomenta y AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    MatMultKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C);
    
    // Espera a que todos terminen
    hipDeviceSynchronize();
    
    // Lee C from del GPU
    hipMemcpy(C.elementos,d_C.elementos,tam,hipMemcpyDeviceToHost);//  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    
    // Libera memoria GPU
    hipFree(d_A.elementos);//  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    hipFree(d_B.elementos);//  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
    hipFree(d_C.elementos);//  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
}

// Toma un elemento de la matriz
__device__ int GetElement(const Matriz A, int ren, int col) {
    return A.elementos[ren* A.salto + col];
}

// Pon un elemento en la matriz
__device__ void SetElement(Matriz A, int ren, int col, int value) {
    A.elementos[ren* A.salto + col] = value;
}

// Toma una submatriz de A de tamaño TAM_BLOQUExTAM_BLOQUE
// localizada col sub-matrices a la derecha y ren sub-matrices abajo
// desde la esquina superior izquierda
__device__ Matriz LeeSubMatriz(Matriz A, int ren, int col) {
    Matriz Asub;
    Asub.nRen = TAM_BLOQUE;
    Asub.nCol = TAM_BLOQUE;
    Asub.salto = A.salto;
    Asub.elementos = &A.elementos[A.salto * TAM_BLOQUE * ren+ TAM_BLOQUE * col];
    return Asub;
}


// Kernel multiplicación de Matriz
__global__ void MatMultKernel(Matriz A, Matriz B, Matriz C) {
    
    // Renglon y columna del bloque
    int blockRen = blockIdx.y;
    int blockCol = blockIdx.x;
    
    // Cada bloque calcula una submatriz Csub de C
    Matriz Csub = LeeSubMatriz(C, blockRen, blockCol);
    
    // Cada thread calcula un elemento de Csub
    // acumulando elementos en valorC
    int valorC= 0;
    
    // Thread ren y col dentro de Csub
    int ren = threadIdx.y;
    int col = threadIdx.x;
    
    // Loop sobre todas las sub-matrices de A y B necesarias
    // para calcular Csub
    // Multiplica cada par de sub-matrices y acumula resultados
    for (int m = 0; m < (A.nRen / TAM_BLOQUE); ++m) {

        // Toma sub-Matriz Asub de A
        Matriz Asub = LeeSubMatriz(A,blockRen,m);//  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
        
        // Toma sub-Matriz Bsub de B
        Matriz Bsub = LeeSubMatriz(B,m,blockCol);//  AGREGA LOS ARGUMENTOS QUE CORRESPONDAN
        
        // La memoria compartida donde se almacenan Asub y Bsub
        __shared__ int As[TAM_BLOQUE][TAM_BLOQUE];
        __shared__ int Bs[TAM_BLOQUE][TAM_BLOQUE];
        
        // Transfiere Asub y Bsub de memoria global a shared
        // Cada thread carga un elemento de cada submatriz
        As[ren][col] = GetElement(Asub, ren, col);
        Bs[ren][col] = GetElement(Bsub, ren, col);
        
        // Punto de sincronización: Espera a que todas las
        // sub-matrices se hayan cargado antes de continuar
        __syncthreads();
        
        // Multiplica Asub y Bsub
        for (int e = 0; e < TAM_BLOQUE; ++e)
            // Descomenta y agrega la operación apropiada
            valorC += As[ren][e]*Bs[e][col];
        
        // Punto de sincronización antes de iniciar otra iteración
        __syncthreads();
    }
    
    // Escribe Csub a memoria global
    // Cada thread escribe un elemento
    SetElement(Csub, ren, col, valorC);
}


int main(int argc, char* argv[]){
    
    clock_t begin=clock();  // Para medir cuánto tarda
    char *verbose;
    if(argc > 2) verbose = argv[2];
    else verbose = NULL;

    Matriz A, B, C;
    int a1, a2, b1, b2;         // Solo matrices cuadradas
    a1 = atoi(argv[1]);         /* nCol de A */
    // a2 = atoi(argv[2]);         /* nRen  de A */
    // b1 = a2;                    /* nCol de B */
    // b2 = atoi(argv[3]);         /* nRen  de B */
    a2 = a1;         /* nRen  de A */
    b1 = a1;                    /* nCol de B */
    b2 = a1;         /* nRen  de B */
    
    if(argc > 2) verbose = argv[2];
    else verbose = NULL;

    A.nCol = a1;
    A.nRen = a2;
    A.elementos = (int*)malloc(A.nRen * A.nCol * sizeof(int));
    
    B.nCol = b1;
    B.nRen = b2;
    B.elementos = (int*)malloc(B.nRen * B.nCol * sizeof(int));
    
    C.nCol = A.nCol;
    C.nRen = B.nRen;
    C.elementos = (int*)malloc(C.nRen * C.nCol * sizeof(int));

    // Llena las matrices con 1's
    for(int i = 0; i < A.nCol; i++)
        for(int j = 0; j < A.nRen; j++)
            // A.elementos[i*A.nRen + j] = (rand() % 3);
            A.elementos[i*A.nRen + j] = 1;
    
    for(int i = 0; i < B.nCol; i++)
        for(int j = 0; j < B.nRen; j++)
            // B.elementos[i*B.nRen + j] = (rand() % 2);
            B.elementos[i*B.nRen + j] = 1;
    
    MatMult(A, B, C);
    
    clock_t end=clock();  // Checa el tiempo inmediatamente después de terminar
    
    double diffticks=end-begin;
    double diffms=(diffticks*10)/CLOCKS_PER_SEC;
    

    // Imprime hasta porciones de 10x10 de las tres matrices
    if(verbose != NULL && verbose[1] == 'v'){
        for(int i = 0; i < min(10, C.nCol); i++){
            for(int j = 0; j < min(10, C.nRen); j++)
                printf("%d ", C.elementos[i*C.nRen + j]);
            printf("\n");
        }
       printf("\n");
   }

    printf("Tiempo usado: %f mSeg\n\n", diffms);
    
}